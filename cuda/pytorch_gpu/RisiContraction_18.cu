#include "hip/hip_runtime.h"
#include <iostream>
#include <cstring>
#include <string>
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <vector>
#include <algorithm>
#include <thread>
#include <assert.h>

#include <torch/torch.h>

using namespace std;

// +----------------------------+
// | Additional Functionalities |
// +----------------------------+

// Number of contractions
const int NCONTRACTIONS = 18;

// Ceiling
int rounded_division(int number1, int number2) {
	if (number1 % number2 == 0) {
		return number1 / number2;
	}
	return number1 / number2 + 1;
}

// +-------------------------------------------+
// | Atomic Addition Operation For Double Type |
// +-------------------------------------------+ 

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
	static __inline__ __device__ double atomicAdd(double *address, double val) {
		unsigned long long int* address_as_ull = (unsigned long long int*) address;
		unsigned long long int old = *address_as_ull, assumed;
		if (val == 0.0) {
			return __longlong_as_double(old);
		}
		do {
			assumed = old;
			old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val +__longlong_as_double(assumed)));
		} while (assumed != old);
		return __longlong_as_double(old);
	}
#endif

// +-------------------------------------+
// | Kernel Function For The Forward Job |
// +-------------------------------------+

__global__ void RisiContraction_18_forward_job(double *tensor, double *adj, double *value, int N, int nChannels) {
	__shared__ int nContractions;
	__shared__ int A;
	__shared__ int B;
	__shared__ int C;
	__shared__ int Y;

	nContractions = NCONTRACTIONS;

	int global_threadId = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (global_threadId < N * N * nChannels * nContractions) {	
		C = nChannels;
		B = N * C;
		A = N * B;

		Y = nChannels * nContractions;
		
		int f = (global_threadId % Y) % nChannels;
		int Case = (global_threadId % Y) / nChannels + 1;
		int y = (global_threadId / Y) % N;
		int x = (global_threadId / Y) / N;

		int a, b, c, d, e;
		double adj_value;

		double sum = 0.0;

		// +-----------+
		// | 1 + 1 + 1 |
		// +-----------+

		// Case 1 (1/50): Fix a, b. Contract c, d, e.
		if (Case == 1) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}
				
		// Case 2 (3/50): Fix a, d. Contract b, c, e.
		if (Case == 2) {		
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (b = 0; b < N; ++b) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}	
		}
		
		// Case 3 (5/50): Fix b, c. Contract a, d, e.
		if (Case == 3) {		
			b = x;
			c = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						for (a = 0; a < N; ++a) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}	
		}

		// Case 4 (6/50): Fix b, d. Contract a, c, e.
		if (Case == 4) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (a = 0; a < N; ++a) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}

		// Case 5 (10/50): Fix d, e. Contract a, b, c.
		if (Case == 5) {		
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (a = 0; a < N; ++a) {
					for (b = 0; b < N; ++b) {
						for (c = 0; c < N; ++c) {
							sum += tensor[a * A + b * B + c * C + f] * adj_value;
						}
					}
				}
			}
		}

		// +-------+
		// | 1 + 2 |
		// +-------+

		// Case 6 (11/50): (a, b). Contract (c, d). Singleton (e).
		if (Case == 6) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					c = d;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}

		// Case 7 (13/50): (a, b). Contract (d, e). Singleton (c).
		if (Case == 7) {
			a = x;
			b = y;

			for (d = 0; d < N; ++d) {
				e = d;
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 8 (17/50): (a, d). Contract (b, c). Singleton (e).
		if (Case == 8) {
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (b = 0; b < N; ++b) {
						c = b;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 9 (18/50): (a, d). Contract (b, e). Singleton (c).
		if (Case == 9) {
			a = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					b = e;
					for (c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 10 (23/50): (b, c). Contract (a, d). Singleton (e).
		if (Case == 10) {
			b = x;
			c = y;

			for (d = 0; d < N; ++d) {
				for (e = 0; e < N; ++e) {
					adj_value = adj[d * N + e];
					if (adj_value > 0) {
						a = d;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 11 (26/50): (b, d). Contract (a, c). Singleton (e).
		if (Case == 11) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					for (a = 0; a < N; ++a) {
						c = a;
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 12 (27/50): (b, d). Contract (a, e). Singleton (c).
		if (Case == 12) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					a = e;
					for (int c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 13 (28/50): (b, d). Contract (c, e). Singleton (a).
		if (Case == 13) {
			b = x;
			d = y;

			for (e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					c = e;
					for (int a = 0; a < N; ++a) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 14 (38/50): (d, e). Contract (a, b). Singleton (c).
		if (Case == 14) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int a = 0; a < N; ++a) {
					b = a;
					for (int c = 0; c < N; ++c) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// Case 15 (40/50): (d, e). Contract (b, c). Singleton (a).
		if (Case == 15) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int b = 0; b < N; ++b) {
					c = b;
					for (int a = 0; a < N; ++a) {
						sum += tensor[a * A + b * B + c * C + f] * adj_value;
					}
				}
			}
		}

		// +---+
		// | 3 |
		// +---+

		// Case 16 (43/50): (a, d). Contract (b, c, e).
		if (Case == 16) {
			a = x;
			d = y;

			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					b = e;
					c = e;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}	

		// Case 17 (46/50): (b, d). Contract (a, c, e).
		if (Case == 17) {
			b = x;
			d = y;

			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];
				if (adj_value > 0) {
					a = e;
					c = e;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}

		// Case 18 (50/50): (d, e). Contract (a, b, c).
		if (Case == 18) {
			d = x;
			e = y;

			adj_value = adj[d * N + e];
			if (adj_value > 0) {
				for (int a = 0; a < N; ++a) {
					b = a;
					c = a;
					sum += tensor[a * A + b * B + c * C + f] * adj_value;
				}
			}
		}

		value[global_threadId] = sum;
	}
}

// +--------------------------------------+
// | Kernel Function For The Backward Job |
// +--------------------------------------+

__global__ void RisiContraction_18_backward_job(double *tensor_gradient, double *adj, double *gradient, int N, int nChannels) {
	
	__shared__ int nContractions;
	__shared__ int X;
	__shared__ int Y;

	nContractions = NCONTRACTIONS;

	int global_threadId = blockIdx.x * blockDim.x + threadIdx.x;

	if (global_threadId < N * N * N * nChannels) {
		X = N * nChannels * nContractions;
		Y = nChannels * nContractions;

		int f = global_threadId % nChannels;
		int c = (global_threadId / nChannels) % N;
		int b = ((global_threadId / nChannels) / N) % N;
		int a = ((global_threadId / nChannels) / N) / N;

		double sum = 0.0;

		int ind;
		double adj_value;

		for (int d = 0; d < N; ++d) {
			for (int e = 0; e < N; ++e) {
				adj_value = adj[d * N + e];

				if (adj_value > 0) {
					// +-----------+
					// | 1 + 1 + 1 |
					// +-----------+

					// Case 1 (1/50): Fix a, b. Contract c, d, e.
					ind = a * X + b * Y + 0 * nChannels + f;
					sum += gradient[ind] * adj_value;

					// Case 2 (3/50): Fix a, d. Contract b, c, e.
					ind = a * X + d * Y + 1 * nChannels + f;
					sum += gradient[ind] * adj_value;

					// Case 3 (5/50): Fix b, c. Contract a, d, e.
					ind = b * X + c * Y + 2 * nChannels + f;
					sum += gradient[ind] * adj_value;

					// Case 4 (6/50): Fix b, d. Contract a, c, e.
					ind = b * X + d * Y + 3 * nChannels + f;
					sum += gradient[ind] * adj_value;

					// Case 5 (10/50): Fix d, e. Contract a, b, c.
					ind = d * X + e * Y + 4 * nChannels + f;
					sum += gradient[ind] * adj_value;

					// +-------+
					// | 1 + 2 |
					// +-------+

					// Case 6 (11/50): (a, b). Contract (c, d). Singleton (e).
					if (c == d) {
						ind = a * X + b * Y + 5 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 7 (13/50): (a, b). Contract (d, e). Singleton (c).
					if (d == e) {
						ind = a * X + b * Y + 6 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 8 (17/50): (a, d). Contract (b, c). Singleton (e).
					if (b == c) {
						ind = a * X + d * Y + 7 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 9 (18/50): (a, d). Contract (b, e). Singleton (c).
					if (b == e) {
						ind = a * X + d * Y + 8 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 10 (23/50): (b, c). Contract (a, d). Singleton (e).
					if (a == d) {
						ind = b * X + c * Y + 9 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 11 (26/50): (b, d). Contract (a, c). Singleton (e).
					if (a == c) {
						ind = b * X + d * Y + 10 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 12 (27/50): (b, d). Contract (a, e). Singleton (c).
					if (a == e) {
						ind = b * X + d * Y + 11 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 13 (28/50): (b, d). Contract (c, e). Singleton (a).
					if (c == e) {
						ind = b * X + d * Y + 12 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 14 (38/50): (d, e). Contract (a, b). Singleton (c).
					if (a == b) {
						ind = d * X + e * Y + 13 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 15 (40/50): (d, e). Contract (b, c). Singleton (a).
					if (b == c) {
						ind = d * X + e * Y + 14 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// +---+
					// | 3 |
					// +---+

					// Case 16 (43/50): (a, d). Contract (b, c, e).
					if ((b == c) && (c == e))  {
						ind = a * X + d * Y + 15 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 17 (46/50): (b, d). Contract (a, c, e).
					if ((a == c) && (c == e))  {
						ind = b * X + d * Y + 16 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}

					// Case 18 (50/50): (d, e). Contract (a, b, c).
					if ((a == b) && (b == c))  {
						ind = d * X + e * Y + 17 * nChannels + f;
						sum += gradient[ind] * adj_value;
					}
				}
			}
		}
		
		tensor_gradient[global_threadId] += sum;
	}
}

#define CHECK_CUDA(x) TORCH_CHECK(x.type().is_cuda(), #x " must be a CUDA tensor")
#define CHECK_CONTIGUOUS(x) TORCH_CHECK(x.is_contiguous(), #x " must be contiguous")
#define CHECK_INPUT(x) CHECK_CUDA(x); CHECK_CONTIGUOUS(x)

void RisiContraction_18_forward(
	const torch::Tensor &tensor,
	const torch::Tensor &adj,
	torch::Tensor &value,
	const int nThreads = 1024
) {
	CHECK_INPUT(tensor);
	CHECK_INPUT(adj);
	CHECK_INPUT(value);

	assert(tensor.dim() == 4);
	assert(adj.dim() == 2);
	assert(value.dim() == 3);

	const int N = tensor.size(0);
	const int nChannels = tensor.size(3);

	assert(tensor.size(1) == N);
	assert(tensor.size(2) == N);
	assert(adj.size(0) == N);
	assert(adj.size(1) == N);
	assert(value.size(0) == N);
	assert(value.size(1) == N);
	assert(value.size(2) == nChannels * NCONTRACTIONS);

	const int size = value.numel();
	dim3 dimGrid(rounded_division(size, nThreads));
	dim3 dimBlock(nThreads);

	// Kernel launch
	RisiContraction_18_forward_job <<< dimGrid, dimBlock >>> (
		tensor.data<double>(), 
		adj.data<double>(), 
		value.data<double>(), 
		N, 
		nChannels);
}

void RisiContraction_18_backward(
	torch::Tensor &tensor_gradient,
	const torch::Tensor &adj,
	const torch::Tensor &value_gradient,
	const int nThreads = 1024
) {
	CHECK_INPUT(tensor_gradient);
	CHECK_INPUT(adj);
	CHECK_INPUT(value_gradient);

	assert(tensor_gradient.dim() == 4);
	assert(adj.dim() == 2);
	assert(value_gradient.dim() == 3);

	const int N = tensor_gradient.size(0);
	const int nChannels = tensor_gradient.size(3);

	assert(tensor_gradient.size(1) == N);
	assert(tensor_gradient.size(2) == N);
	assert(adj.size(0) == N);
	assert(adj.size(1) == N);
	assert(value_gradient.size(0) == N);
	assert(value_gradient.size(1) == N);
	assert(value_gradient.size(2) == nChannels * NCONTRACTIONS);

	const int size = tensor_gradient.numel();
	dim3 dimGrid(rounded_division(size, nThreads));
	dim3 dimBlock(nThreads);

	// Kernel launch
	RisiContraction_18_backward_job <<< dimGrid, dimBlock >>> (
		tensor_gradient.data<double>(), 
		adj.data<double>(), 
		value_gradient.data<double>(), 
		N, 
		nChannels);
}

std::vector<at::Tensor> test_api(const std::vector<at::Tensor> &tensors) {
	const int N = tensors.size();
	std::vector<at::Tensor> result;
	for (int i = 0; i < N; ++i) {
		result.push_back(torch::zeros({}));
	}
	return result;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
	m.def("test_api", &test_api, "Test API");
	m.def("RisiContraction_18_forward", &RisiContraction_18_forward, "Forward functionality");
	m.def("RisiContraction_18_backward", &RisiContraction_18_backward, "Backward functionality");
}
